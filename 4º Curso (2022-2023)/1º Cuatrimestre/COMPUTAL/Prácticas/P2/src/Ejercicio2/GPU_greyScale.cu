#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <math.h>
#include <sys/time.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define NCHANNELS 4

__global__ void changeRGB_toGrey(uint8_t *im, int width, int height, int nchannels, uint8_t *grey_image)
{
    unsigned char *offset = im + (threadIdx.x + blockDim.x * blockIdx.x) * nchannels;
    int r = offset[0];
    int g = offset[1];
    int b = offset[2];
    grey_image[threadIdx.x + blockDim.x * blockIdx.x] = (int)(0.2989 * r + 0.5870 * g + 0.1140 * b);
}

int main(int nargs, char **argv)
{
    int width, height, nchannels;
    uint8_t *d_rgb_image, *d_grey_image;
    struct timeval fin,ini;

    if (nargs < 2)
    {
        printf("Usage: %s <image1> [<image2> ...]\n", argv[0]);
    }
    // For each image
    // Bucle 0
    for (int file_i = 1; file_i < nargs; file_i++)
    {
        printf("[info] Processing %s\n", argv[file_i]);
        /****** Reading file ******/
        uint8_t *rgb_image = stbi_load(argv[file_i], &width, &height, &nchannels, NCHANNELS);
        if (!rgb_image)
        {
            perror("Image could not be opened");
        }

        /****** Allocating memory ******/
        // - RGB2Grey
        uint8_t *grey_image = (uint8_t *)malloc(width * height);
        if (!grey_image)
        {
            perror("Could not allocate memory");
        }

        // Alloc space for device copy
        hipMalloc((uint8_t **)&d_rgb_image, width*height*NCHANNELS);
        hipMalloc((uint8_t **)&d_grey_image, width*height);

        // Copy to device
        hipMemcpy(d_rgb_image, rgb_image, width*height*NCHANNELS, hipMemcpyHostToDevice);

        // - Filenames 
        for (int i = strlen(argv[file_i]) - 1; i >= 0; i--)
        {
            if (argv[file_i][i] == '.')
            {
                argv[file_i][i] = 0;
                break;
            }
        }

        char *grey_image_filename = 0;
        asprintf(&grey_image_filename, "%s_GPUgrey.jpg", argv[file_i]);
        if (!grey_image_filename)
        {
            perror("Could not allocate memory");
            exit(-1);
        }

        /****** Computations ******/
        printf("[info] %s: width=%d, height=%d, nchannels=%d\n", argv[file_i], width, height, nchannels);

        if (nchannels != 3 && nchannels != 4)
        {
            printf("[error] Num of channels=%d not supported. Only three (RGB), four (RGBA) are supported.\n", nchannels);
            continue;
        }

        // RGB to grey scale
        gettimeofday(&ini,NULL);
        changeRGB_toGrey<<<width,height>>>(d_rgb_image, width, height, NCHANNELS, d_grey_image);
        gettimeofday(&fin,NULL);

        hipMemcpy(grey_image, d_grey_image, width*height, hipMemcpyDeviceToHost);

        stbi_write_jpg(grey_image_filename, width, height, 1, grey_image, 10);
        free(rgb_image);

	      printf("Tiempo: %f\n", ((fin.tv_sec*1000000+fin.tv_usec)-(ini.tv_sec*1000000+ini.tv_usec))*1.0/1000000.0);
        free(grey_image_filename);
    }
}
