#include <hip/hip_runtime.h>

#include <stdio.h>
#include <algorithm>
#include <iostream>
#include <sys/time.h>
using namespace std;
#define RADIUS 3
//#define BLOCK_SIZE 16
int N = 16;
int BLOCK_SIZE = 16;


__global__ void stencil_1d(int *in, int *out, int N, int BLOCK_SIZE) {
  extern __shared__ int temp[/*BLOCK_SIZE + 2 * RADIUS*/];
  int gindex = threadIdx.x + blockIdx.x * blockDim.x;
  int lindex = threadIdx.x + RADIUS;


  // Read input elements into shared memory
  //solucionamos desbordes cuando n/block_size da un decimal
  if(gindex>=N) temp[lindex] = in[N-1]; else
  temp[lindex] = in[gindex];
  if (threadIdx.x < RADIUS) {
    
    // Resolvemos el problema de índices fuera de rango
    int in_index_ini = gindex - RADIUS;
    if(in_index_ini < 0){ //si desborda por la izq
      in_index_ini = 0;
    }

    int in_index_fin = gindex + BLOCK_SIZE;
    if(in_index_fin >= N){ //si desborda por la derecha
      in_index_fin = N-1;
    }

    temp[lindex - RADIUS] = in[in_index_ini];
    temp[lindex + BLOCK_SIZE] = in[in_index_fin];
  }
  
  // Synchronize (ensure all the data is available)
  __syncthreads();
  
  // Apply the stencil
  int result = 0;
    for (int offset = -RADIUS ; offset <= RADIUS ; offset++){
      result += temp[lindex + offset];
    }
  if(gindex>N)
    return;
  // Store the result
  out[gindex] = result;
}

void fill_ints(int *x, int n) {
  fill_n(x, n, 1);
}

int main(int argc, char *argv[]) {
  int N = 16;
  //int BLOCK_SIZE = 16;
  int size;
  int *in, *out;      // host copies of a, b, c
  int *d_in, *d_out;  // device copies of a, b, c
  struct timeval ini, fin;

  if(argc < 2){
    printf("insuficientes argumentos: ./exec N BLOCK_SIZE");
    return 0;
  }
  else{
    N = atoi(argv[1]);
    BLOCK_SIZE = atoi(argv[2]);
  }

  size = N * sizeof(int);

  // Alloc space for host copies and setup values
  in = (int *)malloc(size); fill_ints(in, N);
  out = (int *)malloc(size); fill_ints(out, N);

  // Alloc space for device copies
  hipMalloc((void **)&d_in, size);
  hipMalloc((void **)&d_out, size);

  // Copy to device
  hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
  hipMemcpy(d_out, out, size, hipMemcpyHostToDevice);

  // Launch stencil_1d() kernel on GPU
  gettimeofday(&ini,NULL);

  if(N%BLOCK_SIZE==0)
    stencil_1d<<<N/BLOCK_SIZE,BLOCK_SIZE, (BLOCK_SIZE + 2 * RADIUS)*sizeof(int)>>>(d_in, d_out, N, BLOCK_SIZE);
  else
    stencil_1d<<<N/BLOCK_SIZE+1,BLOCK_SIZE, (BLOCK_SIZE + 2 * RADIUS)*sizeof(int)>>>(d_in, d_out, N, BLOCK_SIZE);

  gettimeofday(&fin,NULL);

  printf("%d %f\n", N, ((fin.tv_sec*1000000+fin.tv_usec)-(ini.tv_sec*1000000+ini.tv_usec))*1.0/1000000.0);
  
  // Copy result back to host
  hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
/* debug
printf("\n");
  for (int i=0; i!=N; i++){
    printf(" %d", out[i]);
  }*/

  // Cleanup
  free(in); free(out);
  hipFree(d_in); hipFree(d_out);
  return 0;
}
